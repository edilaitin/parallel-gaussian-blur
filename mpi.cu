#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <stdint.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>

#include "mpi.h"

#define IMAGESIZE 54
// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                                     \
    do                                                                                           \
    {                                                                                            \
        hipError_t cuErr = call;                                                                \
        if (hipSuccess != cuErr)                                                                \
        {                                                                                        \
            printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr)); \
            exit(0);                                                                             \
        }                                                                                        \
    } while (0)

#pragma pack(push, 2)
typedef struct
{
    char sign;
    int size;
    int notused;
    int data;
    int headwidth;
    int width;
    int height;
    short numofplanes;
    short bitpix;
    int method;
    int arraywidth;
    int horizresol;
    int vertresol;
    int colnum;
    int basecolnum;
} img;
#pragma pop

unsigned char *openImg(int inputFileNumber, img *bmp);
void generateImg(unsigned char *imgdata, img *bmp);
int setBoundary(int i, int min, int max);

__global__ void perform(int nStart, int nStop, int width, int height, int radius, unsigned char *red, unsigned char *green, unsigned char *blue)
{
    printf("HERE");
    int t = threadIdx.x;
    for (int i = nStart; i < nStop; i++)
    {
        for (int j = 0; j < width; j++)
        {
            double row;
            double col;
            double redSum = 0;
            double greenSum = 0;
            double blueSum = 0;
            double weightSum = 0;
            for (row = i - radius; row <= i + radius; row++)
            {
                for (col = j - radius; col <= j + radius; col++)
                {
                    int x = col;
                    if (x < 0)
                        x = 0;
                    else if (x > width - 1)
                        x = width - 1;
                    int y = row;
                    if (y < 0)
                        y = 0;
                    else if (y > height - 1)
                        y = height - 1;
                    int tempPos = y * width + x;
                    double square = (col - j) * (col - j) + (row - i) * (row - i);
                    double sigma = radius * radius;
                    double weight = exp(-square / (2 * sigma)) / (3.14 * 2 * sigma);
                    redSum += red[tempPos] * weight;
                    greenSum += green[tempPos] * weight;
                    blueSum += blue[tempPos] * weight;
                    weightSum += weight;
                }
            }
            red[i * width + j] = round(redSum / weightSum);
            green[i * width + j] = round(greenSum / weightSum);
            blue[i * width + j] = round(blueSum / weightSum);
            redSum = 0;
            greenSum = 0;
            blueSum = 0;
            weightSum = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    unsigned char *imgdata;
    img *bmp = (img *)malloc(IMAGESIZE);
    int radius = atoi(argv[1]);
    int inputFileNumber = atoi(argv[2]);
    imgdata = openImg(inputFileNumber, bmp);

    int width = bmp->width;
    int height = bmp->height;
    int SIZE = width * height * sizeof(unsigned char);

    int i, j;
    int rgb_width = width * 3;
    if ((width * 3 % 4) != 0)
    {
        rgb_width += (4 - (width * 3 % 4));
    }

    unsigned char *red;
    unsigned char *green;
    unsigned char *blue;
    cudaErrorCheck(hipMallocManaged(&red, width * height * sizeof(unsigned char)));
    cudaErrorCheck(hipMallocManaged(&green, width * height * sizeof(unsigned char)));
    cudaErrorCheck(hipMallocManaged(&blue, width * height * sizeof(unsigned char)));

    int pos = 0;
    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width * 3; j += 3, pos++)
        {
            red[pos] = imgdata[i * rgb_width + j];
            green[pos] = imgdata[i * rgb_width + j + 1];
            blue[pos] = imgdata[i * rgb_width + j + 2];
        }
    }

    struct timeval start_time, stop_time, elapsed_time;
    gettimeofday(&start_time, NULL);

    int my_PE_num;
    int threadNumber;
    int nStart, nStop;

    MPI_Status status;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_PE_num);
    MPI_Comm_size(MPI_COMM_WORLD, &threadNumber);

    // Map MPI ranks to GPUs
    // cudaErrorCheck(hipSetDevice(my_PE_num));
    // int deviceCount = 0;
    // hipGetDeviceCount(&deviceCount);
    // printf("%d", deviceCount);

    int subSize = height / threadNumber;

    nStart = my_PE_num * subSize;
    nStop = (my_PE_num + 1) * subSize;

    unsigned char *redBuffer;
    unsigned char *greenBuffer;
    unsigned char *blueBuffer;
    cudaErrorCheck(hipMallocManaged(&redBuffer, width * height * sizeof(unsigned char)));
    cudaErrorCheck(hipMallocManaged(&greenBuffer, width * height * sizeof(unsigned char)));
    cudaErrorCheck(hipMallocManaged(&blueBuffer, width * height * sizeof(unsigned char)));

    if (my_PE_num == 0)
    {
        int k, n_proc;
        MPI_Comm_size(MPI_COMM_WORLD, &n_proc);
        for (k = 1; k < n_proc; k++)
        {
            MPI_Recv(redBuffer, SIZE, MPI_UNSIGNED_CHAR, k, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MPI_Recv(greenBuffer, SIZE, MPI_UNSIGNED_CHAR, k, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
            MPI_Recv(blueBuffer, SIZE, MPI_UNSIGNED_CHAR, k, MPI_ANY_TAG, MPI_COMM_WORLD, &status);

            int otherThreadStart = k * subSize;
            int otherThreadStop = (k + 1) * subSize;

            for (i = otherThreadStart; i < otherThreadStop; i++)
            {
                for (j = 0; j < width; j++)
                {
                    red[i * width + j] = redBuffer[i * width + j];
                    green[i * width + j] = greenBuffer[i * width + j];
                    blue[i * width + j] = blueBuffer[i * width + j];
                }
            }
        }
        perform<<<1, 10>>>(nStart, nStop, width, height, radius, red, green, blue);
        cudaErrorCheck(hipPeekAtLastError());
        cudaErrorCheck(hipDeviceSynchronize());
    }

    else
    {
        perform<<<1, 10>>>(nStart, nStop, width, height, radius, red, green, blue);
        cudaErrorCheck(hipPeekAtLastError());
        cudaErrorCheck(hipDeviceSynchronize());
        for (i = nStart; i < nStop; i++)
        {
            for (j = 0; j < width; j++)
            {
                redBuffer[i * width + j] = red[i * width + j];
                greenBuffer[i * width + j] = green[i * width + j];
                blueBuffer[i * width + j] = blue[i * width + j];
            }
        }

        MPI_Send(redBuffer, SIZE, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(greenBuffer, SIZE, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(blueBuffer, SIZE, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD);
    }

    if (my_PE_num == 0)
    {
        // print elapsed time
        gettimeofday(&stop_time, NULL);
        timersub(&stop_time, &start_time, &elapsed_time);
        printf("Took %f seconds \n", elapsed_time.tv_sec + elapsed_time.tv_usec / 1000000.0);

        pos = 0;
        for (i = 0; i < height; i++)
        {
            for (j = 0; j < width * 3; j += 3, pos++)
            {
                imgdata[i * rgb_width + j] = red[pos];
                imgdata[i * rgb_width + j + 1] = green[pos];
                imgdata[i * rgb_width + j + 2] = blue[pos];
            }
        }
        generateImg(imgdata, bmp);
    }
    cudaErrorCheck(hipFree(redBuffer));
    cudaErrorCheck(hipFree(greenBuffer));
    cudaErrorCheck(hipFree(blueBuffer));

    MPI_Finalize();
    hipFree(red);
    hipFree(green);
    hipFree(blue);
    free(bmp);
    return 0;
}

unsigned char *openImg(int inputFileNumber, img *in)
{
    char inPutFileNameBuffer[32];
    sprintf(inPutFileNameBuffer, "%d.bmp", inputFileNumber);

    FILE *file;
    if (!(file = fopen(inPutFileNameBuffer, "rb")))
    {
        printf("File not found!");
        free(in);
        exit(1);
    }
    fread(in, 54, 1, file);

    unsigned char *data = (unsigned char *)malloc(in->arraywidth);
    fseek(file, in->data, SEEK_SET);
    fread(data, in->arraywidth, 1, file);
    fclose(file);
    return data;
}

void generateImg(unsigned char *imgdata, img *out)
{
    FILE *file;
    time_t now;
    time(&now);
    char fileNameBuffer[32];
    sprintf(fileNameBuffer, "%s.bmp", ctime(&now));
    file = fopen(fileNameBuffer, "wb");
    fwrite(out, IMAGESIZE, 1, file);
    fseek(file, out->data, SEEK_SET);
    fwrite(imgdata, out->arraywidth, 1, file);
    fclose(file);
}